#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>

const int KERNEL_SIZE = 5;

__global__ void MedianFilter(const int rows, const int cols, const double *data, double *data_result) {

    unsigned int xIdx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int yIdx = threadIdx.y + blockIdx.y * blockDim.y;
    const int b = KERNEL_SIZE / 2;
    const int n = KERNEL_SIZE * KERNEL_SIZE;

    double pix, tmp;
    double arr[n] = {0};
    int counter = 0, i, j;

    if (xIdx > b && yIdx > b && yIdx < (rows - b) && xIdx < (cols - b)) {
        for (int k = -b; k <= b; k++) {
            for (int l = -b; l <= b; l++) {
                arr[counter] = data[(yIdx + l) * cols + (xIdx + k)];
                counter++;
            }
        }

        // Сортировка пузырьком, простите :)
        for (i = 0; i < n; i++) {
            for (j = i + 1; j < n; j++) {
                if (arr[i] > arr[j]) {
                    tmp = arr[i];
                    arr[i] = arr[j];
                    arr[j] = tmp;
                }
            }
        }

        pix = arr[b];
    } else {
        pix = data[yIdx * cols + xIdx];
    }
    data_result[yIdx * cols + xIdx] = pix;
}


int main() {

    cv::Mat image = cv::imread("../dog_noise.png", cv::IMREAD_GRAYSCALE);
    uchar *data_char = image.data;
    int rows = image.rows;
    int cols = image.cols;
    unsigned long N = rows * cols;
    double *h_data = new double[N];
    double *h_data_result = new double[N];
    double *d_data;
    double *d_data_result;

    hipMalloc(&d_data, N * sizeof(double));
    hipMalloc(&d_data_result, N * sizeof(double));


    for (int i = 0; i < N; i++) {
        h_data[i] = data_char[i] / 255.0;
    }

    const dim3 blockSize(8, 8, 1);
    const dim3 gridSize(cols / blockSize.x + 1, rows / blockSize.y + 1, 1);

    hipMemcpy(d_data, h_data, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_data_result, h_data_result, N * sizeof(double), hipMemcpyHostToDevice);

    MedianFilter<<<gridSize, blockSize>>>(rows, cols, d_data, d_data_result);

    hipMemcpy(h_data_result, d_data_result, N * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        h_data_result[i] = h_data_result[i] * 255.0;
    }

    cv::Mat imgGray(rows, cols, CV_64F, h_data_result);

    cv::imwrite("../dog_median_filter.jpg", imgGray);

    delete[] h_data;
    delete[] h_data_result;
    hipFree(d_data);
    hipFree(d_data_result);

    return 0;
}
