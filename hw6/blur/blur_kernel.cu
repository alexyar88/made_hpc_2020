#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>


__global__ void Blur(int kernel_size, int rows, int cols, const double *data, double *data_result) {
    unsigned int xIdx = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned int yIdx = threadIdx.y + blockIdx.y * blockDim.y;
    int b = kernel_size / 2;
    int n = kernel_size * kernel_size;

    double pix = 0;
    if (xIdx > b && yIdx > b && yIdx < (rows - b) && xIdx < (cols - b)) {
        for (int k = -b; k <= b; k++) {
            for (int l = -b; l <= b; l++) {
                pix += (1.0 / n) * data[(yIdx + l) * cols + (xIdx + k)];
            }
        }
    } else {
        pix = data[yIdx * cols + xIdx];
    }
    data_result[yIdx * cols + xIdx] = pix;
}

int main() {
    int kernel_size = 15;

    cv::Mat image = cv::imread("../dog.png", cv::IMREAD_GRAYSCALE);
    uchar *data_char = image.data;
    int rows = image.rows;
    int cols = image.cols;
    unsigned long N = rows * cols;
    double *h_data = new double[N];
    double *h_data_result = new double[N];
    double *d_data;
    double *d_data_result;

    hipMalloc(&d_data, N * sizeof(double));
    hipMalloc(&d_data_result, N * sizeof(double));


    for (int i = 0; i < N; i++) {
        h_data[i] = data_char[i] / 255.0;
    }

    const dim3 blockSize(8, 8, 1);
    const dim3 gridSize(cols / blockSize.x + 1, rows / blockSize.y + 1, 1);

    hipMemcpy(d_data, h_data, N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_data_result, h_data_result, N * sizeof(double), hipMemcpyHostToDevice);

    Blur<<<gridSize, blockSize>>>(kernel_size, rows, cols, d_data, d_data_result);

    hipMemcpy(h_data_result, d_data_result, N * sizeof(double), hipMemcpyDeviceToHost);

    for (int i = 0; i < N; i++) {
        h_data_result[i] = h_data_result[i] * 255.0;
    }

    cv::Mat imgGray(rows, cols, CV_64F, h_data_result);

    cv::imwrite("../dog_result.png", imgGray);

    delete [] h_data;
    delete [] h_data_result;
    hipFree(d_data);
    hipFree(d_data_result);

    return 0;
}
